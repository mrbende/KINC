
#include <hip/hip_runtime.h>






extern "C" __device__ float Pearson_computeCluster(
   const float2 *data,
   const char *labels,
   int sampleSize,
   char cluster,
   int minSamples)
{
   // compute intermediate sums
   int n = 0;
   float sumx = 0;
   float sumy = 0;
   float sumx2 = 0;
   float sumy2 = 0;
   float sumxy = 0;

   for ( int i = 0, j = 0; i < sampleSize; ++i )
   {
      if ( labels[i] >= 0 )
      {
         if ( labels[i] == cluster )
         {
            float x_i = data[j].x;
            float y_i = data[j].y;

            sumx += x_i;
            sumy += y_i;
            sumx2 += x_i * x_i;
            sumy2 += y_i * y_i;
            sumxy += x_i * y_i;

            ++n;
         }

         ++j;
      }
   }

   // compute correlation only if there are enough samples
   float result = NAN;

   if ( n >= minSamples )
   {
      result = (n*sumxy - sumx*sumy) / sqrt((n*sumx2 - sumx*sumx) * (n*sumy2 - sumy*sumy));
   }

   return result;
}






extern "C" __global__ void Pearson_compute(
   const float2 *in_data,
   char clusterSize,
   const char *in_labels,
   int sampleSize,
   int minSamples,
   float *out_correlations)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   const float2 *data = &in_data[i * sampleSize];
   const char *labels = &in_labels[i * sampleSize];
   float *correlations = &out_correlations[i * clusterSize];

   for ( char k = 0; k < clusterSize; ++k )
   {
      correlations[k] = Pearson_computeCluster(data, labels, sampleSize, k, minSamples);
   }
}
