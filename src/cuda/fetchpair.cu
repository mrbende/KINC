#include "hip/hip_runtime.h"

// #include "linalg.cu"






extern "C" __global__ void fetchPair(
   const float *expressions,
   int sampleSize,
   const int2 *in_index,
   int minExpression,
   Vector2 *out_X,
   int *out_N,
   char *out_labels)
{
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   // initialize variables
   int2 index = in_index[i];
   Vector2 *X = &out_X[i * sampleSize];
   char *labels = &out_labels[i * sampleSize];
   int *p_numSamples = &out_N[i];

   if ( index.x == 0 && index.y == 0 )
   {
      return;
   }

   // index into gene expressions
   const float *gene1 = &expressions[index.x * sampleSize];
   const float *gene2 = &expressions[index.y * sampleSize];

   // populate X with shared expressions of gene pair
   int numSamples = 0;

   for ( int i = 0; i < sampleSize; ++i )
   {
      if ( isnan(gene1[i]) || isnan(gene2[i]) )
      {
         labels[i] = -9;
      }
      else if ( gene1[i] < minExpression || gene2[i] < minExpression )
      {
         labels[i] = -6;
      }
      else
      {
         X[numSamples] = make_float2(gene1[i], gene2[i]);
         numSamples++;

         labels[i] = 0;
      }
   }

   // return size of X
   *p_numSamples = numSamples;
}
